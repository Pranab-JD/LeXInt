#include "hip/hip_runtime.h"
#include <fstream>
#include <string>
#include <sstream>
#include <sys/types.h>
#include <sys/stat.h>

#include "../Timer.hpp"

//? Problems
#include "Diff_Adv_2D.hpp"
#include "Burgers_2D.hpp"

//! ---------------------------------------------------------------------------

//! Include Exponential Integrators and Leja functions 
//! (This has to be included to use Leja and/or exponential integrators)
#include "../Leja_GPU.hpp"

//! Functions to compute the largest eigenvalue (in magnitude)
#include "../Eigenvalues.hpp"

//! ---------------------------------------------------------------------------

using namespace std;

//? ====================================================================================== ?//

//! Read Leja points from file
vector<double> Leja_Points()
{
    int max_Leja_pts = 1000;                        // Max. number of Leja points
    vector<double> Leja_X(max_Leja_pts);            // Initialize static array
    int count = 0;                                  // Loop counter variable

    //* Load Leja points
    ifstream inputFile;
    inputFile.open("../Leja_10000.txt");

    //* Read Leja_points from file into the vector Leja_X
    while(count < max_Leja_pts && inputFile >> Leja_X[count])
    {
        count = count + 1;
    }

    inputFile.close();

    return Leja_X;
}

//? ====================================================================================== ?//

int main()
{
    //* Initialise parameters
    int n = 2048;                                   // # grid points
    int N = n*n;                                    // # grid points
    double xmin = -1;                               // Left boundary (limit)
    double xmax =  1;                               // Right boundary (limit)
    double ymin = -1;                               // Left boundary (limit)
    double ymax =  1;                               // Right boundary (limit)
    vector<double> X(n);                            // Array of grid points
    vector<double> Y(n);                            // Array of grid points
    vector<double> u(N);                            // Initial condition

    //* Set up X, Y arrays and initial condition
    for (int ii = 0; ii < n; ii++)
    {
        X[ii] = xmin + ii*(xmax - xmin)/n;
        Y[ii] = ymin + ii*(ymax - ymin)/n;
    }

    //* Initialise additional parameters
    double dx = X[12] - X[11];                              // Grid spacing
    double dy = Y[12] - Y[11];                              // Grid spacing
    double velocity = 50;                                  // Advection speed
    double dif_cfl = (dx*dx * dy*dy)/(2*dx*dx + 2*dy*dy);   // Diffusion CFL
    double adv_cfl = dx*dy/(velocity * (dx + dy));          // Advection CFL
    double dt = 2*min(dif_cfl, adv_cfl);                  // Step size
    stringstream step_size;
    step_size << fixed << scientific << setprecision(1) << dt;
    cout << endl << "Step size: " << dt << endl;

    //* Temporal parameters
    double time = 0;                                        // Simulation time elapsed
    double t_final = 0.001;                                  // Final simulation time
    int time_steps = 0;                                     // # time steps

    //* Set of Leja points
    vector<double> Leja_X = Leja_Points();

    //? Choose problem and integrator
    double tol = 1e-10;
    string problem = "Diff_Adv_2D";
    string integrator = "Hom_Linear";

    RHS_Dif_Adv_2D RHS(n, dx, dy, velocity);                //* Default problem
    Leja_GPU<RHS_Dif_Adv_2D> leja_gpu{N, integrator};       //* Default problem

    //! Error Check
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    if (problem == "Diff_Adv_2D")
    {
        //? Initial condition
        for (int ii = 0; ii < n; ii++)
        {
            for (int jj = 0; jj< n; jj++)
            {
                u[n*ii + jj] = 1 + exp(-((X[ii] + 0.5)*(X[ii] + 0.5) + (Y[jj] + 0.5)*(Y[jj] + 0.5))/0.01);
            }
        }
    }
    else if (problem == "Burgers_2D")
    {
        RHS_Burgers_2D RHS(n, dx, dy, velocity);
        Leja_GPU<RHS_Burgers_2D> leja_gpu{N, integrator};

        //? Initial condition
        for (int ii = 0; ii < n; ii++)
        {
            for (int jj = 0; jj< n; jj++)
            {
                u[n*ii + jj] = 2 + 0.01*sin(2*M_PI*X[ii]) + 0.01*sin(8*M_PI*X[ii] + 0.3)
                                 + 0.01*sin(2*M_PI*Y[jj]) + 0.01*sin(8*M_PI*Y[jj] + 0.3);
            }
        }
    }
    else
    {
        cout << "Undefined problem!" << endl;
    } 

    hipDeviceSynchronize();
    gpuErrchk( hipPeekAtLastError() );

    //! Allocate memory on GPU
    size_t N_size = N * sizeof(double);
    double *device_u; hipMalloc(&device_u, N_size);
    double *device_u_low; hipMalloc(&device_u_low, N_size);
    double *device_u_sol; hipMalloc(&device_u_sol, N_size);
    double *device_error; hipMalloc(&device_error, N_size);
    double *device_auxillary_Leja; hipMalloc(&device_auxillary_Leja, N_size);
    double *device_auxillary_Jv; hipMalloc(&device_auxillary_Jv, 7*N_size);        //* To compute spectrum using power iterations
    hipMemcpy(device_u, &u[0], N_size, hipMemcpyHostToDevice);                    //* Copy state variable to device

    //! Set GPU spport to true
    bool GPU_access = true;
    GPU_handle hipblas.h;

    //? Shifting and scaling parameters
    double eigenvalue = 0.0;
    LeXInt::Power_iterations(RHS, device_u, N, eigenvalue, device_auxillary_Jv, GPU_access, hipblas.h);         // Real eigenvalue has to be negative
    eigenvalue = -1.2*eigenvalue;
    double c = eigenvalue/2.0; double Gamma = -eigenvalue/4.0;
    cout << "Largest eigenvalue: " << eigenvalue << endl;

    //! Create nested directories (for movies)
    // int sys_value = system(("mkdir -p ../../LeXInt_Test/DA_GPU/"));
    // string directory = "../../LeXInt_Test/DA_GPU/";

    //! Error Check
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    //! Time Loop
    LeXInt::timer time_loop;
    time_loop.start();

    while (time < t_final)
    {
        //* Final time step
        if (time + dt >= t_final)
        {
            dt = t_final - time;
        }

        hipDeviceSynchronize();

        //? ---------------------------------------------------------------- ?//

        //? Homogenous Linear Equations

        if (integrator == "Hom_Linear")
        {
            LeXInt::real_Leja_exp(RHS, device_u, device_u_sol, device_auxillary_Leja, N, Leja_X, c, Gamma, tol, dt, GPU_access, hipblas.h);
        }
        
        //? ---------------------------------------------------------------- ?//

        //? Nonlinear Equations

        //* Non-embedded Intergators
        else if (integrator == "Rosenbrock_Euler" or integrator == "EPIRK4s3B")
        {
            // * ----------- Eigenvalue (Spectrum) ----------- *//

            if (time_steps % 100 == 0)
            {
                //? Largest eigenvalue of the Jacobian; changes at every time step for nonlinear equations
                eigenvalue = 0.0;
                LeXInt::Power_iterations(RHS, device_u, N, eigenvalue, device_auxillary_Jv, GPU_access, hipblas.h);         // Real eigenvalue has to be negative
                eigenvalue = -1.2*eigenvalue;
                c = eigenvalue/2.0; Gamma = -eigenvalue/4.0;
                cout << "Largest eigenvalue: " << eigenvalue << endl;
            }

            //* ---------------------------------------------- *//

            //? Embedded integrators
            leja_gpu(RHS, device_u, device_u_sol, N, Leja_X, c, Gamma, tol, dt, GPU_access);
        }

        //* Embedded Integrators 
        else if (integrator == "EXPRB32" or integrator == "EXPRB42" or integrator == "EXPRB43" or integrator == "EXPRB53s3" 
        or integrator == "EXPRB54s4" or integrator == "EPIRK4s3" or integrator == "EPIRK4s3A" or integrator == "EPIRK5P1")
        {
            // * ----------- Eigenvalue (Spectrum) ----------- *//

            if (time_steps % 100 == 0)
            {
                //? Largest eigenvalue of the Jacobian; changes at every time step for nonlinear equations
                double eigenvalue = 0.0;
                LeXInt::Power_iterations(RHS, device_u, N, eigenvalue, device_auxillary_Jv, GPU_access, hipblas.h);         // Real eigenvalue has to be negative
                eigenvalue = -1.2*eigenvalue;
                double c = eigenvalue/2.0; double Gamma = -eigenvalue/4.0;
                cout << "Largest eigenvalue: " << eigenvalue << endl;
            }

            //* ---------------------------------------------- *//

            //? Embedded integrators
            leja_gpu(RHS, device_u, device_u_low, device_u_sol, N, Leja_X, c, Gamma, tol, dt, GPU_access);
            
            LeXInt::axpby(1.0, device_u_low, -1.0, device_u_sol, device_error, N, GPU_access);
            double error = LeXInt::l2norm(device_error, N, GPU_access, hipblas.h);
            cout << "Embedded error: " << error << endl;
        }
        else
        {
            cout << "ERROR: Please choose an available integator. See 'Leja.hpp'." << endl;
        }

        //? ---------------------------------------------------------------- ?//

        //* Update variables
        time = time + dt;
        LeXInt::copy(device_u_sol, device_u, N, GPU_access);
        time_steps = time_steps + 1;

        if (time_steps % 1000 == 0)
        {
            cout << "Time steps: " << time_steps << endl;
            cout << "Time elapsed: " << time << endl;
            cout << endl;
        }

        //! Write data to files (for movies)
        // string output_data = directory + "/" +  to_string(time_steps) + ".txt";
        // ofstream data;
        // data.open(output_data); 
        // hipMemcpy(&u[0], device_u_sol, N_size, hipMemcpyDeviceToHost);   
        // for(int ii = 0; ii < N; ii++)
        // {
        //     data << setprecision(16) << u[ii] << endl;
        // }
        // data.close();
    }

    time_loop.stop();

    //* Copy state variable from device to host
    hipMemcpy(&u[0], device_u, N_size, hipMemcpyDeviceToHost);                

    cout << endl << "==================================================" << endl;
    cout << "Simulation time: " << time << endl;
    cout << "Total number of time steps: " << time_steps << endl;
    cout << "Total time elapsed (s): " << time_loop.total() << endl;
    cout << "==================================================" << endl << endl;

    //! Create nested directories
    int sys_value_f = system(("mkdir -p ../../LeXInt_Test/" + to_string(GPU_access) + "/Constant/" + problem + "/dt_" + step_size.str()).c_str());
    string directory_f = "../../LeXInt_Test/" + to_string(GPU_access) + "/Constant/" + problem + "/dt_" + step_size.str();

    //? Write data to files
    string final_data = directory_f + "/Final_data.txt";
    ofstream data;
    data.open(final_data);
    for(int ii = 0; ii < N; ii++)
    {
        data << setprecision(16) << u[ii] << endl;
    }
    data.close();

    string results = directory_f + "/Results.txt";
    ofstream params;
    params.open(results);
    params << "Simulation time: " << time << endl;
    params << "Total number of time steps: " << time_steps << endl;
    params << setprecision(16) << "Total time elapsed (s): " << time_loop.total() << endl;
    params.close();

    cout << "Writing data to files complete!" << endl << endl;

    return 0;
}